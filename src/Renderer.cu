#include "hip/hip_runtime.h"
#include "Renderer.cuh"
#include <iostream>
#include "cuda-wrapper/cuda.cuh"

#include "Window.cuh"
#include "Color.cuh"
#include "Camera.cuh"
#include "Ray.cuh"
#include "Hittable.cuh"
#include "Scene.cuh"
#include "Vec3.cuh"
#include <hiprand/hiprand_kernel.h>

#define COLOR_NORMALS true

Renderer::Renderer()
{
}

Renderer::~Renderer()
{
}

__device__ float map(float input, float input_start, float input_end, float output_start, float output_end)
{
    return output_start + (output_end - output_start) * ((input - input_start) / (input_end - input_start));
}

__device__ Point random_in_unit_sphere(hiprandState *local_rand_state)
{
    while (true)
    {
        Vec3 point = Vec3(hiprand_uniform(local_rand_state), hiprand_uniform(local_rand_state), hiprand_uniform(local_rand_state)) * 2.0f - Vec3(1, 1, 1);
        if (point.magnitude_squared() >= 1)
            continue;
        return point;
    }
}

__device__ Direction random_unit_vector(hiprandState *local_rand_state)
{
    return random_in_unit_sphere(local_rand_state).normalize();
}
__device__ Point random_in_hemisphere(const Direction &normal, hiprandState *local_rand_state)
{
    Point in_unit_sphere = random_in_unit_sphere(local_rand_state);
    if (in_unit_sphere.dot(normal) > 0.0)
    {
        return in_unit_sphere;
    }
    else
    {
        return -in_unit_sphere;
    }
}

__device__ FloatColor trace_ray(Ray &ray, Camera &camera, Scene *scene, hiprandState local_rand_state)
{
    Ray current_ray = Ray{ray.direction, ray.origin};
    FloatColor current_attenuation = FloatColor{1.0f, 1.0f, 1.0f};

    for (int _ = 0; _ < 50; _++)
    {
        Hit closest_hit = Hit{INFINITY, Vec3(0, 0, 0), nullptr};

        for (int i = 0; i < scene->sphere_count; i++)
        {
            Hit hit;
            Sphere *sphere = (scene->spheres + i);
            hit = sphere->hit(current_ray);

            if (hit.t > 0.001 && hit.t < camera.far && hit.t < closest_hit.t)
            {
                closest_hit = hit;
            }
        }

        if (closest_hit.hittable)
        {
            Sphere sphere = *((Sphere *)closest_hit.hittable);
            // Direction normal = sphere.position - closest_hit.p;
            Direction normal = closest_hit.p - sphere.position;
            // normal.x = -normal.x;
            // normal.y = -normal.y;
            // normal.z = -normal.z;

            normal = normal.normalize();
            if (COLOR_NORMALS)
            {
                return FloatColor{normal.x + 1, normal.y + 1, normal.z + 1} * 0.5;
                // normal = (normal + 1) * 0.5;
                // normal = -normal + 1;
                // return FloatColor{normal.x, normal.y, normal.z};
            }

            current_attenuation = current_attenuation * sphere.get_material().color;
            current_ray.direction = closest_hit.p + normal + random_in_unit_sphere(&local_rand_state);
            // current_ray.direction = closest_hit.p + normal + random_in_hemisphere(normal, &local_rand_state);
            // current_ray.direction = closest_hit.p + normal + random_unit_vector(&local_rand_state);

            current_ray.origin = closest_hit.p;
        }
        else
        {
            Direction ray_unit_direction = current_ray.direction.normalize();
            float y = 0.5 * (ray_unit_direction.y + 1.0);
            FloatColor background_color = FloatColor(1.0, 1.0, 1.0) * (1.0 - y) + FloatColor(0.5, 0.7, 1.0) * y;
            // FloatColor background_color = FloatColor(1, 1, 1);
            return background_color * current_attenuation;

            // return FloatColor(1.0, 1.0, 1.0) * (1.0f - darkness) + (FloatColor(0.5, .7, 1.0) * (darkness));
            // return camera.background * current_attenuation;
        }
    }

    // max bounces reached
    return FloatColor{1, 0, 0};
}

__global__ void gpuRender(uint32_t *sampled_pixels, int pixel_count, size_t image_width, size_t image_height, Scene *scene, hiprandState *rand_state, int seed)
{
    int x = (threadIdx.x + blockIdx.x * blockDim.x);
    int y = (threadIdx.y + blockIdx.y * blockDim.y);

    if (x >= image_width || y >= image_height)
        return;

    int index = ((image_height - y - 1) * image_width + (x)) * 3;

    hiprand_init(seed, index, 0, &rand_state[index / 3]);
    hiprandState local_rand_state = rand_state[index / 3];
    float random_x = (hiprand_uniform(&local_rand_state));
    float random_y = (hiprand_uniform(&local_rand_state));

    // printf("random_x: %f, random_y: %f\n", random_x, random_y);
    // sampled_pixels[index] += 255;
    // sampled_pixels[index + 1] += 255;
    // sampled_pixels[index + 2] += 255;
    // scene->camera.to_viewport(x, y);
    // camera.to_viewport(x, y);
    Camera camera = scene->camera;
    Direction ray_direction = scene->camera.to_viewport(x + random_x, y + random_y) - camera.position;
    Ray ray = {.direction = ray_direction, .origin = camera.position};

    // return;
    Color color = trace_ray(ray, camera, scene, local_rand_state).square_root().to_int_color();

    sampled_pixels[index] += color.r;
    sampled_pixels[index + 1] += color.g;
    sampled_pixels[index + 2] += color.b;
}

void Renderer::render(uint32_t *h_sampled_pixels, Scene *scene, Window &window, int seed)
{

    size_t pixels_size = window.image_width * window.image_height * 3 * sizeof(uint32_t);
    size_t pixel_count = window.image_height * window.image_width;

    hiprandState *d_rand_state = (hiprandState *)cuda::malloc(pixel_count * sizeof(hiprandState));

    // Set up device memory
    uint32_t *d_pixels = (uint32_t *)cuda::malloc(pixels_size);
    cuda::copyToDevice(d_pixels, h_sampled_pixels, pixels_size);

    int tx = 16;
    int ty = 16;

    dim3 blocks(window.image_width / tx + 1, window.image_height / ty + 1);
    dim3 threads(tx, ty);
    gpuRender<<<blocks, threads>>>(d_pixels, pixel_count, window.image_width, window.image_height, scene, d_rand_state, seed);
    cuda::synchronize();

    cuda::copyToHost(h_sampled_pixels, d_pixels, pixels_size);
    cuda::free(d_pixels);
    cuda::free(d_rand_state);
}
